#include "hip/hip_runtime.h"
/*
 * This file is part of Vlasiator.
 * Copyright 2010-2016 Finnish Meteorological Institute
 *
 * For details of usage, see the COPYING file and read the "Rules of the Road"
 * at http://www.physics.helsinki.fi/vlasiator/
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "cuda_moments_kernel.cuh"
#include "../cuda_context.cuh"
#include "../common.h"

#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

using namespace std;

MomentInfo *dev_momentInfos[MAXCPUTHREADS];
MomentInfo *host_momentInfos[MAXCPUTHREADS];

Real *dev_momentArrays1[MAXCPUTHREADS];
Real *host_momentArrays1[MAXCPUTHREADS];
Real *dev_momentArrays2[MAXCPUTHREADS];
Real *host_momentArrays2[MAXCPUTHREADS];

bool isCudaMomentsAllocated = false;

__host__ void cuda_allocateMomentCalculations(
   const uint nPopulations,
   const uint maxThreads
   ) {
   if (isCudaMomentsAllocated) return;
   for (uint cpuThreadID=0; cpuThreadID<maxThreads; ++cpuThreadID) {

      //hipMalloc
      HANDLE_ERROR( hipMalloc((void**)&dev_momentInfos[cpuThreadID], nPopulations*sizeof(MomentInfo)) );
      HANDLE_ERROR( hipMalloc((void**)&dev_momentArrays1[cpuThreadID], CUDABLOCKS*nMoments1*(nPopulations+1)*sizeof(Real)) );
      HANDLE_ERROR( hipMalloc((void**)&dev_momentArrays2[cpuThreadID], CUDABLOCKS*nMoments2*(nPopulations+1)*sizeof(Real)) );

      // Also allocate and pin memory on host for faster transfers
      HANDLE_ERROR( hipHostAlloc((void**)&host_momentInfos[cpuThreadID], nPopulations*sizeof(MomentInfo), hipHostMallocPortable) );
      HANDLE_ERROR( hipHostAlloc((void**)&host_momentArrays1[cpuThreadID], CUDABLOCKS*nMoments1*(nPopulations+1)*sizeof(Real), hipHostMallocPortable) );
      HANDLE_ERROR( hipHostAlloc((void**)&host_momentArrays2[cpuThreadID], CUDABLOCKS*nMoments2*(nPopulations+1)*sizeof(Real), hipHostMallocPortable) );
   }
   isCudaMomentsAllocated = true;
   return;
}


// Define kernel for calculating zeroth and first velocity moments
__global__ void moments_first_kernel(
   MomentInfo *dev_momentInfos,
   Real* dev_momentArrays1,
   const int nPopulations
   ){

   const int cudaBlocks = gridDim.x;
   const int blocki = blockIdx.x;
   const int i = threadIdx.x;
   const int j = threadIdx.y;
   const int k = threadIdx.z;
   const uint ti = k*WID2 + j*WID + i;

   const Real HALF = 0.5;
   __shared__ Real n_sum[WID3];
   __shared__ Real nvx_sum[WID3];
   __shared__ Real nvy_sum[WID3];
   __shared__ Real nvz_sum[WID3];

   const uint offset = blocki*nMoments1*(nPopulations+1);
   if (ti==0) {
      dev_momentArrays1[offset + nPopulations*nMoments1 + 0] = 0;
      dev_momentArrays1[offset + nPopulations*nMoments1 + 1] = 0;
      dev_momentArrays1[offset + nPopulations*nMoments1 + 2] = 0;
      dev_momentArrays1[offset + nPopulations*nMoments1 + 3] = 0;
      dev_momentArrays1[offset + nPopulations*nMoments1 + 4] = 0;
   }
   
   for (uint popID=0; popID<nPopulations; ++popID) {
      n_sum[ti] = 0.0;
      nvx_sum[ti] = 0.0;
      nvy_sum[ti] = 0.0;
      nvz_sum[ti] = 0.0;

      const uint nBlocks = dev_momentInfos[popID].blockCount;
      const Real mass = dev_momentInfos[popID].mass;
      const Real charge = dev_momentInfos[popID].charge;
      Real* blockParams = dev_momentInfos[popID].parameterPointer;
      const Real DV3 = blockParams[BlockParams::DVX]*blockParams[BlockParams::DVY]*blockParams[BlockParams::DVZ];

      for (uint blockLID=blocki; blockLID<nBlocks; blockLID += cudaBlocks) {
         if (blockLID >= nBlocks) break;
         Real* blockParams = dev_momentInfos[popID].parameterPointer + blockLID*BlockParams::N_VELOCITY_BLOCK_PARAMS;
         Realf* avgs = dev_momentInfos[popID].meshDataPointer +blockLID*WID3;
         const Real VX = blockParams[BlockParams::VXCRD] + (i+HALF)*blockParams[BlockParams::DVX];
         const Real VY = blockParams[BlockParams::VYCRD] + (j+HALF)*blockParams[BlockParams::DVY];
         const Real VZ = blockParams[BlockParams::VZCRD] + (k+HALF)*blockParams[BlockParams::DVZ];

         n_sum[ti]   += avgs[ti];
         nvx_sum[ti] += avgs[ti]*VX;
         nvy_sum[ti] += avgs[ti]*VY;
         nvz_sum[ti] += avgs[ti]*VZ;
      }
      __syncthreads();
      // Implemented just a simple non-optimized thread sum
      for (unsigned int s=WID3/2; s>0; s>>=1) {
         if (ti < s) {
            n_sum[ti] += n_sum[ti + s];
            nvx_sum[ti] += nvx_sum[ti + s];
            nvy_sum[ti] += nvy_sum[ti + s];
            nvz_sum[ti] += nvz_sum[ti + s];
         }
         __syncthreads();
      }
      if (ti==0) {
         dev_momentArrays1[offset + popID*nMoments1 + 0] = n_sum[0]   * DV3;
         dev_momentArrays1[offset + popID*nMoments1 + 1] = nvx_sum[0] * DV3;
         dev_momentArrays1[offset + popID*nMoments1 + 2] = nvy_sum[0] * DV3;
         dev_momentArrays1[offset + popID*nMoments1 + 3] = nvz_sum[0] * DV3;

         // Sum over all populations
         dev_momentArrays1[offset + nPopulations*nMoments1 + 0] += n_sum[0]   * DV3 * mass;
         dev_momentArrays1[offset + nPopulations*nMoments1 + 1] += nvx_sum[0] * DV3 * mass;
         dev_momentArrays1[offset + nPopulations*nMoments1 + 2] += nvy_sum[0] * DV3 * mass;
         dev_momentArrays1[offset + nPopulations*nMoments1 + 3] += nvz_sum[0] * DV3 * mass;
         dev_momentArrays1[offset + nPopulations*nMoments1 + 4] += n_sum[0]   * DV3 * charge;
      }
   }
   return;
}

// Define kernel for calculating second velocity moments
__global__ void moments_second_kernel(
   MomentInfo *dev_momentInfos,
   Real* dev_momentArrays2,
   const int nPopulations,
   const Real bulkVX,
   const Real bulkVY,
   const Real bulkVZ
   ){

   const int cudaBlocks = gridDim.x;
   const int blocki = blockIdx.x;
   const int i = threadIdx.x;
   const int j = threadIdx.y;
   const int k = threadIdx.z;
   const uint ti = k*WID2 + j*WID + i;

   const Real HALF = 0.5;
   __shared__ Real nvx2_sum[WID3];
   __shared__ Real nvy2_sum[WID3];
   __shared__ Real nvz2_sum[WID3];

   const uint offset = blocki*nMoments2*(nPopulations+1);
   if (ti==0) {
      dev_momentArrays2[offset + nPopulations*nMoments2 + 0] = 0;
      dev_momentArrays2[offset + nPopulations*nMoments2 + 1] = 0;
      dev_momentArrays2[offset + nPopulations*nMoments2 + 2] = 0;
   }

   for (uint popID=0; popID<nPopulations; ++popID) {
      nvx2_sum[ti] = 0.0;
      nvy2_sum[ti] = 0.0;
      nvz2_sum[ti] = 0.0;

      const uint nBlocks = dev_momentInfos[popID].blockCount;
      const Real mass = dev_momentInfos[popID].mass;
      //const Real charge = dev_momentInfos[popID].charge;
      Real* blockParams = dev_momentInfos[popID].parameterPointer;
      const Real DV3 = blockParams[BlockParams::DVX]*blockParams[BlockParams::DVY]*blockParams[BlockParams::DVZ];

      for (uint blockLID=blocki; blockLID<nBlocks; blockLID += cudaBlocks) {
         if (blockLID >= nBlocks) break;
         Real* blockParams = dev_momentInfos[popID].parameterPointer + blockLID*BlockParams::N_VELOCITY_BLOCK_PARAMS;
         Realf* avgs = dev_momentInfos[popID].meshDataPointer +blockLID*WID3;
         const Real VX = blockParams[BlockParams::VXCRD] + (i+HALF)*blockParams[BlockParams::DVX];
         const Real VY = blockParams[BlockParams::VYCRD] + (j+HALF)*blockParams[BlockParams::DVY];
         const Real VZ = blockParams[BlockParams::VZCRD] + (k+HALF)*blockParams[BlockParams::DVZ];

         nvx2_sum[ti] += avgs[ti] * (VX - bulkVX) * (VX - bulkVX);
         nvy2_sum[ti] += avgs[ti] * (VY - bulkVY) * (VY - bulkVY);
         nvz2_sum[ti] += avgs[ti] * (VZ - bulkVZ) * (VZ - bulkVZ);
      }

      __syncthreads();
      // Implemented just a simple non-optimized thread sum
      for (unsigned int s=WID3/2; s>0; s>>=1) {
         if (ti < s) {
            nvx2_sum[ti] += nvx2_sum[ti + s];
            nvy2_sum[ti] += nvy2_sum[ti + s];
            nvz2_sum[ti] += nvz2_sum[ti + s];
         }
         __syncthreads();
      }
      if (ti==0) {
         dev_momentArrays2[offset + popID*nMoments2 + 0] = nvx2_sum[0] * DV3 * mass;
         dev_momentArrays2[offset + popID*nMoments2 + 1] = nvy2_sum[0] * DV3 * mass;
         dev_momentArrays2[offset + popID*nMoments2 + 2] = nvz2_sum[0] * DV3 * mass;

         // Sum over all populations
         dev_momentArrays2[offset + nPopulations*nMoments2 + 0] += nvx2_sum[0] * DV3 * mass;
         dev_momentArrays2[offset + nPopulations*nMoments2 + 1] += nvy2_sum[0] * DV3 * mass;
         dev_momentArrays2[offset + nPopulations*nMoments2 + 2] += nvz2_sum[0] * DV3 * mass;
      }
   }
   return;
}

// Define kernel caller glue
void calculate_firstMoments_glue(
   MomentInfo *dev_momentInfos,
   Real* dev_momentArrays1,
   const int nPopulations,
   hipStream_t stream
   ) {
   dim3 block(WID,WID,WID);
   moments_first_kernel<<<CUDABLOCKS, block, 4*WID3*sizeof(Real), stream>>> (
      dev_momentInfos,
      dev_momentArrays1,
      nPopulations);
   return;
}
void calculate_secondMoments_glue(
   MomentInfo *dev_momentInfos,
   Real* dev_momentArrays2,
   const int nPopulations,
   const Real bulkVX,
   const Real bulkVY,
   const Real bulkVZ,
   hipStream_t stream
   ) {
   dim3 block(WID,WID,WID);
   moments_second_kernel<<<CUDABLOCKS, block, 3*WID3*sizeof(Real), stream>>> (
      dev_momentInfos,
      dev_momentArrays2,
      nPopulations,
      bulkVX,
      bulkVY,
      bulkVZ);
   return;
}
